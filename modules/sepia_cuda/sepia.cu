#include "hip/hip_runtime.h"
#include "sepia.h"
#include <math.h>
#include <malloc.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <plugin.h>
#include <string.h>


extern "C" void __init_sepia_cuda(int* result, plugin_action_info_t *init_data) {
	if(init_data == NULL || result == NULL)
	{ 
		*result = 1;
		return;
	}
	init_data->name = (char*)malloc(3*sizeof(char));
	strcpy(init_data->name, "-scuda");
	init_data->func_ptr = (void*)&sepia;
	init_data->instruction = (char*)"sepia effect based on CUDA";
	init_data->argc = 0;
	init_data->type = FUNC_TRANSFORM;
	*result = 0;
}

__global__ void cuda_func(pixel_t* pixels) {
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;
	static const float c[3][3] = {
		{ .393f, .769f, .189f },
		{ .349f, .686f, .168f },
		{ .272f, .543f, .131f } };
	struct pixel_t old = pixels[i];
	
	int r = old.r * c[0][0] + old.g * c[0][1] + old.b * c[0][2];
	int g = old.r * c[1][0] + old.g * c[1][1] + old.b * c[1][2];
	int b = old.r * c[2][0] + old.g * c[2][1] + old.b * c[2][2];
	
	pixels[i].r = (unsigned char)min(255, r);
	pixels[i].g = (unsigned char)min(255, g);
	pixels[i].b = (unsigned char)min(255, b);
}

/*static struct pixel_t setPixel(struct image_t* const image, int x, int y) {
	static const float c[3][3] = {
		{ .393f, .769f, .189f },
		{ .349f, .686f, .168f },
		{ .272f, .543f, .131f } };
	struct pixel_t const old = image->pixels[y * (image->width) + x];
	struct pixel_t pixel;
	
	pixel.r = sat(old.r * c[0][0] + old.g * c[0][1] + old.b * c[0][2]);
	pixel.g = sat(old.r * c[1][0] + old.g * c[1][1] + old.b * c[1][2]);
	pixel.b = sat(old.r * c[2][0] + old.g * c[2][1] + old.b * c[2][2]);
	return  pixel;
}*/

bmp_transform_error_code_t
sepia(struct image_t* const src, struct image_t* const result, char** argv) {
	size_t N = src->width*src->height;
	clock_t begin, end;
	double time_spent;
	struct pixel_t* dev_pixels, dev_new;
	result->width = src->width;
	result->height = src->height;
	result->pixels = (struct pixel_t*)malloc(sizeof(struct pixel_t) * N);
	begin = clock();
	hipMalloc(&dev_pixels,  N*sizeof(struct pixel_t));
	hipMemcpy(dev_pixels, src->pixels, N*sizeof(struct pixel_t), hipMemcpyHostToDevice);
	cuda_func<<<(N+255) / 256, 256>>>(dev_pixels);
	hipMemcpy(result->pixels, dev_pixels, N*sizeof(struct pixel_t), hipMemcpyDeviceToHost);
	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("CUDA sepia time: %f\n", time_spent);
	return TRANSFORM_OK;
}
